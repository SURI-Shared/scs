#include "hip/hip_runtime.h"
#include "cones.h"

#define BOX_CONE_MAX_ITERS (25)
#define POW_CONE_TOL (1e-9)
#define POW_CONE_MAX_ITERS (20)

/* Box cone limits (+ or -) taken to be INF */
#define MAX_BOX_VAL (1e15)

/*
 * CUDA Routine to scale the limits of one entry in the box cone by the scaling diagonal mat D > 0
 *
 *  want (t, s) \in K <==> (t', s') \in K'
 *
 *  (t', s') = (d0 * t, D s) (overloading D to mean D[1:])
 *    (up to scalar scaling factor which we can ignore due to conic prooperty)
 *
 *   K = { (t, s) | t * l <= s <= t * u, t >= 0 } =>
 *       { (t, s) | d0 * t * D l / d0 <= D s <= d0 * t D u / d0, t >= 0 } =>
 *       { (t', s') | t' * l' <= s' <= t' u', t >= 0 } = K'
 *  where l' = D l  / d0, u' = D u / d0.
 */
void cuda_normalize_box_cone(scs_int j, ScsCone *k, scs_float *D, scs_int bsize) {
  if (j < bsize - 1){
    if (k->bu[j] >= MAX_BOX_VAL) {
      k->bu[j] = INFINITY;
    } else {
      k->bu[j] = D ? D[j + 1] * k->bu[j] / D[0] : k->bu[j];
    }
    if (k->bl[j] <= -MAX_BOX_VAL) {
      k->bl[j] = -INFINITY;
    } else {
      k->bl[j] = D ? D[j + 1] * k->bl[j] / D[0] : k->bl[j];
    }
  }
}

void cuda_scale_box_cone(ScsCone *k, ScsConeWork *c, ScsScaling *scal) {
  if (k->bsize && k->bu && k->bl) {
    c->box_t_warm_start = 1.;
    if (scal) {
      /* also does some sanitizing */
      scs_int j = blockIdx.x*blockDim.x+threadIdx.x;
      cuda_normalize_box_cone(j,k, &(scal->D[k->z + k->l]), k->bsize);
    }
  }
}

__global__ void cuda_proj_box_cone_grad_hess_kernel(const scs_float *bl,const scs_float* bu,const scs_int bsize,const scs_float t,const scs_float* x, const scs_float* rho, scs_float* gt_ele, scs_float* ht_ele){
  scs_int j = threadIdx.x;
  if (j<bsize-1){
    scs_float r = rho ? 1.0 / rho[j] : 1.;
    if (x[j] > t * bu[j]) {
      gt_ele[j] = r * (t * bu[j] - x[j]) * bu[j]; /* gradient */
      ht_ele[j] = r * bu[j] * bu[j];              /* hessian */
    } else if (x[j] < t * bl[j]) {
      gt_ele[j] = r * (t * bl[j] - x[j]) * bl[j]; /* gradient */
      ht_ele[j] = r * bl[j] * bl[j];              /* hessian */
    }
  }
  //accumulate gradient and hessian into first element of arrays
  for(uint stride=(bsize-1)/2;stride>0;stride>>=1){
    if (j<stride){
      gt_ele[j]+=gt_ele[j+stride];
      ht_ele[j]+=ht_ele[j+stride];
    }
    __syncthreads();
  }
}


/* Project onto { (t, s) | t * l <= s <= t * u, t >= 0 }, Newton's method on t
   tx = [t; s], total length = bsize, under Euclidean metric 1/r_box.
   Using a single CUDA thread
*/
static scs_float cuda_proj_box_cone(scs_float *tx, const scs_float *bl,
                               const scs_float *bu, scs_int bsize,
                               scs_float t_warm_start, scs_float *r_box) {
  scs_float *x, gt, ht, t_prev, t = t_warm_start;
  scs_float rho_t = 1, *rho = SCS_NULL, r;
  scs_int iter, j;

  if (bsize == 1) { /* special case */
    tx[0] = MAX(tx[0], 0.0);
    return tx[0];
  }
  x = &(tx[1]);

  if (r_box) {
    rho_t = 1.0 / r_box[0];
    rho = &(r_box[1]);
  }

  /* should only require about 5 or so iterations, 1 or 2 if warm-started */
  scs_float *gt_dev,*ht_dev;
  hipMalloc(&gt_dev,(bsize-1)*sizeof(scs_float));
  hipMalloc(&ht_dev,(bsize-1)*sizeof(scs_float));
  for (iter = 0; iter < BOX_CONE_MAX_ITERS; iter++) {
    t_prev = t;
    gt = rho_t * (t - tx[0]); /* gradient */
    ht = rho_t;               /* hessian */
    cuda_proj_box_cone_grad_hess_kernel<<<1,256>>>(bl,bu,bsize,t,x,rho,gt_dev,ht_dev);//TODO: how should the various cone parameters get to the device?
    gt+=gt_dev[0];
    ht+=ht_dev[0];
    t = MAX(t - gt / MAX(ht, 1e-8), 0.); /* newton step */
#if VERBOSITY > 3
    scs_printf("iter %i, t_new %1.3e, t_prev %1.3e, gt %1.3e, ht %1.3e\n", iter,
               t, t_prev, gt, ht);
    scs_printf("ABS(gt / (ht + 1e-6)) %.4e, ABS(t - t_prev) %.4e\n",
               ABS(gt / (ht + 1e-6)), ABS(t - t_prev));
#endif
    /* TODO: sometimes this check can fail (ie, declare convergence before it
     * should) if ht is very large, which can happen with some pathological
     * problems.
     */
    if (ABS(gt / MAX(ht, 1e-6)) < 1e-12 * MAX(t, 1.) ||
        ABS(t - t_prev) < 1e-11 * MAX(t, 1.)) {
      break;
    }
  }
  hipFree(gt_dev);
  hipFree(ht_dev);
  if (iter == BOX_CONE_MAX_ITERS) {
    scs_printf("warning: box cone proj hit maximum %i iters\n", (int)iter);
  }
  for (j = 0; j < bsize - 1; j++) {
    if (x[j] > t * bu[j]) {
      x[j] = t * bu[j];
    } else if (x[j] < t * bl[j]) {
      x[j] = t * bl[j];
    }
    /* x[j] unchanged otherwise */
  }
  tx[0] = t;

#if VERBOSITY > 3
  scs_printf("box cone iters %i\n", (int)iter + 1);
#endif
  return t;
}

/* project onto SOC of size q using a single CUDA thread*/
void cuda_proj_soc(scs_float *x, scs_int q) {
  if (q == 0) {
    return;
  }
  if (q == 1) {
    x[0] = MAX(x[0], 0.);
    return;
  }
  scs_float v1 = x[0];
  scs_float s = 0;
  for (int i=1;i<q;i++){
    s+=x[i]*x[i];
  }
  s=SQRTF(s);
  scs_float alpha = (s + v1) / 2.0;

  if (s <= v1) {
    return;
  } else if (s <= -v1) {
    memset(&(x[0]), 0, q * sizeof(scs_float));
  } else {
    x[0] = alpha;
    for(int i=1;i<q;i++){
      x[i]*=alpha / s;
    }
  }
}

static void proj_power_cone(scs_float *v, scs_float a) {
  scs_float xh = v[0], yh = v[1], rh = ABS(v[2]);
  scs_float x = 0.0, y = 0.0, r;
  scs_int i;
  /* v in K_a */
  if (xh >= 0 && yh >= 0 &&
      POW_CONE_TOL + POWF(xh, a) * POWF(yh, (1 - a)) >= rh) {
    return;
  }

  /* -v in K_a^* */
  if (xh <= 0 && yh <= 0 &&
      POW_CONE_TOL + POWF(-xh, a) * POWF(-yh, 1 - a) >=
          rh * POWF(a, a) * POWF(1 - a, 1 - a)) {
    v[0] = v[1] = v[2] = 0;
    return;
  }

  r = rh / 2;
  for (i = 0; i < POW_CONE_MAX_ITERS; ++i) {
    scs_float f, fp, dxdr, dydr;
    x = pow_calc_x(r, xh, rh, a);
    y = pow_calc_x(r, yh, rh, 1 - a);

    f = pow_calc_f(x, y, r, a);
    if (ABS(f) < POW_CONE_TOL) {
      break;
    }

    dxdr = pow_calcdxdr(x, xh, rh, r, a);
    dydr = pow_calcdxdr(y, yh, rh, r, (1 - a));
    fp = pow_calc_fp(x, y, dxdr, dydr, a);

    r = MAX(r - f / fp, 0);
    r = MIN(r, rh);
  }
  v[0] = x;
  v[1] = y;
  v[2] = (v[2] < 0) ? -(r) : (r);
}

/* project onto the primal K cone in the paper */
/* the r_y vector determines the INVERSE metric, ie, project under the
 * diag(r_y)^-1 norm.
 */
scs_int cuda_proj_cone(scs_int cone_index, scs_float *x, const ScsCone *k, ScsConeWork *c,
                         scs_int normalize, scs_float *r_y) {
  scs_int done=0;
  scs_int vector_index=0;
  scs_int cone_count=0;

  scs_float *r_box = SCS_NULL;

  if (cone_index<k->z) { /* doesn't use r_y */
    /* project onto primal zero / dual free cone */
    x[cone_index]=0;
    done=1;
  }
  else{
    vector_index=k->z;
    cone_count=k->z;
  }
  if (!done && cone_index<cone_count+k->l) { /* doesn't use r_y */
    /* project onto positive orthant */
    x[cone_index] = MAX(x[cone_index], 0.0);
    done=1;
  }
  else{
    vector_index+=k->l;
    cone_index+=k->l;
  }
  if (!done && cone_index<cone_count+k->bsize) { /* DOES use r_y */
    if (r_y) {
      r_box = &(r_y[cone_index]);
    }
    /* project onto box cone */
    c->box_t_warm_start = cuda_proj_box_cone(&(x[cone_index]), k->bl, k->bu, k->bsize,
                                        c->box_t_warm_start, r_box);
    done=1;
  }
  else{
    vector_index += k->bsize; /* since b = (t,s), len(s) = bsize - 1 */
    cone_count+= k->bsize;
  }
  if (!done && k->q && cone_index<cone_count+k->qsize) { /* doesn't use r_y */
    /* project onto second-order cones */

    //figure out what index in the vectors this cone's entries begin at
    scs_int SOC_cone_index=cone_index-cone_count;//current cone is this index into the SOC cones
    for(scs_int j=0;j<SOC_cone_index;k++){
      vector_index+=k->q[j];
    }
    cuda_proj_soc(&(x[vector_index]),k->q[SOC_cone_index]);
    done=1;
  }
  else{
    for(scs_int j=0;j<k->qsize;k++){//accumulate SOC entries if any
      vector_index+=k->q[j];
    }
    cone_count+=k->qsize;
  }
  if (!done && k->s && cone_index<cone_count+k->ssize) { /* doesn't use r_y */
    /* project onto PSD cones */

    //figure out what index in the vectors this cone's entries begin at
    scs_int PSD_cone_index=cone_index-cone_count;//current cone is this index into the PSD cones
    for(scs_int j=0;j<PSD_cone_index;k++){
      vector_index+=get_sd_cone_size(k->s[j]);
    }
    cuda_proj_semi_definite_cone(&(x[vector_index]),k->s[PSD_cone_index],c);
    done=1;
  }
  else{
    for(scs_int j=0;j<k->ssize;k++){
      vector_index+=get_sd_cone_size(k->s[j]);
    }
    cone_count+=k->ssize;
  }
  if (!done && (k->ep || k->ed) && cone_index< cone_count+k->ep+k->ed ) { /* doesn't use r_y */
    scs_int EXP_cone_index=cone_index-cone_count;
      /* provided in exp_cone.c */
    SCS(proj_pd_exp_cone)(&(x[vector_index + 3 * EXP_cone_index]), EXP_cone_index < k->ep);
    done=1;
  }
  else{
    vector_index += 3 * (k->ep + k->ed);
    cone_count+=k->ep+k->ed;
  }
  if (!done && cone_index<cone_count+k->psize && k->p) { /* doesn't use r_y */
    scs_float v[3];
    scs_int idx;
    scs_int PWR_cone_index=cone_index-cone_count;
    idx = vector_index + 3 * PWR_cone_index;
    if (k->p[PWR_cone_index] >= 0) {
      /* primal power cone */
      proj_power_cone(&(x[idx]), k->p[PWR_cone_index]);
    } else {
      /* dual power cone, using Moreau */
      v[0] = -x[idx];
      v[1] = -x[idx + 1];
      v[2] = -x[idx + 2];

      proj_power_cone(v, -k->p[PWR_cone_index]);

      x[idx] += v[0];
      x[idx + 1] += v[1];
      x[idx + 2] += v[2];
    }
    done=1;
  }else{
    vector_index += 3 * k->psize;
    cone_count+=k->psize;
  }
  /* project onto OTHER cones */
  return 0;
}

/* CUDA Kernel for cone projection routine, performs projection in-place.
   If normalize > 0 then will use normalized (equilibrated) cones if applicable.

   Moreau decomposition for R-norm projections:

    `x + R^{-1} \Pi_{C^*}^{R^{-1}} ( - R x ) = \Pi_C^R ( x )`

   where \Pi^R_C is the projection onto C under the R-norm:

    `||x||_R = \sqrt{x ' R x}`.

*/
__global__
void _cuda_proj_dual_cone_kernel(scs_float *x, ScsConeWork *c, ScsScaling *scal,
                            scs_float *r_y) {
  int status, i;
  ScsCone *k = c->k;

  if (!c->scaled_cones) {
    cuda_scale_box_cone(k, c, scal);
    c->scaled_cones = 1;
  }

  /* copy s = x */
  i=blockIdx.x*blockDim.x+threadIdx.x;
  c->s[i]=x[i];

  /* x -> - Rx */
  x[i] *= r_y ? -r_y[i] : -1;

  /* project -x onto cone, x -> \Pi_{C^*}^{R^{-1}}(-x) under r_y metric */
  status = cuda_proj_cone(i,x, k, c, scal ? 1 : 0, r_y);

  /* return x + R^{-1} \Pi_{C^*}^{R^{-1}} ( -x )  */
  if (r_y) {
    x[i] = x[i] / r_y[i] + c->s[i];
  } else {
    x[i] += c->s[i];
  }
}

int _cuda_proj_dual_cone_host(float *x, ScsConeWork *c, ScsScaling *scal,
                            float *r_y) {
    
}